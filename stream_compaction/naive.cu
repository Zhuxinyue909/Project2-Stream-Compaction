#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include<iostream>
int *dev_A;
int *dev_B;
int *dev_C;
namespace StreamCompaction {
namespace Naive {

// TODO: __global__

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
__global__ void Nscan(int n, int logn, int *Ain, int*Bout,int *Ctemp){//in;out;temp

	int thid = threadIdx.x;
	int offset; 
	Bout[0] = 0;
	Ctemp[0] = 0;
	for (int j = 0; j < n-1; j++){
		Ctemp[j + 1] = Ain[j];
	}

	//Ctemp[thid] = (thid > 0) ? Ain[thid - 1] : 0;
	for (int d = 1; d <= logn; d++){
		offset = 2;

		if (d == 1)offset = 1;
		if (d == 2)offset = 2;
		else 
			for (int i = 1; i < d-1; i++){
				offset *= 2; 
			}
		if (thid >= offset)//pow(2,d-1){d=1,off=1}{d=2,off=2}{d=3,off=4}off=pow(2,d-1){d=4,offset=8}
				Ctemp[thid] += Ctemp[thid - offset];
	}
	Bout[0] = 0;
    Bout[thid] = Ctemp[thid];

	}
void init(int *hst_A, int *hst_B,int n){

		int _size = n *sizeof(int);
		hipMalloc((void**)&dev_A, _size);
		hipMemcpy(dev_A, hst_A, _size, hipMemcpyHostToDevice);

		hipMalloc((void**)&dev_B, _size);
		hipMemcpy(dev_B, hst_B, _size, hipMemcpyHostToDevice);

		hipMalloc((void**)&dev_C, _size);
	}
void scan(int n, int *odata, const int *idata) {
    // TODO
	int num;
	if (n % 2 != 0)
	{
		num = ilog2ceil(n);
		num = pow(2, num);
	}
	else num = n;
	int *_idata=new int[num];
	for (int i = 0; i < num; i++){
		_idata[i] = idata[i];
	}
	init(_idata, odata, num);
	
	//std::cout << ilog2ceil(4) << ilog2ceil(5);//2,3;	
	int logn = ilog2ceil(num);
	Nscan <<< 1, num >> >(num,logn,dev_A, dev_B,dev_C);

	hipMemcpy(odata, dev_B, num* sizeof(int), hipMemcpyDeviceToHost);//destination,source,
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	
    printf("2.1");
}

}
}
