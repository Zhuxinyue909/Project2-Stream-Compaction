#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
    // TODO use `thrust::exclusive_scan`
    // example: for device_vectors dv_in and dv_out:
    // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

	thrust::device_vector<int> dv_in,dv_out;
	for (int i = 0; i < n; i++){
		dv_in.push_back(idata[i]);
		dv_out.push_back(0);
	}
	thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
	thrust::copy(dv_out.begin(), dv_out.end(), odata);


	printf("4.1");

		
}



}

}

