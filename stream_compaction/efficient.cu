#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>
int *dev_A1;
int *dev_B1;

namespace StreamCompaction {
namespace Efficient {

// TODO: __global__

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
__global__ void Uscan(int p1, int p2, int *od){
		int thid = threadIdx.x*2*p1;
		//od[thid + p1 - 1] = id[thid + p1 - 1];
		//od[thid + p2 - 1] = id[thid + p2 - 1];

		od[thid + p2 - 1] += od[thid + p1 - 1];
		
	}
__global__ void put0(int * odata, int n)
	{
				odata[n - 1] = 0;
			}

__global__ void Dscan(int p1,int p2,int *od){

		int thid = threadIdx.x*2*p1;
			
		//od[thid + p1 - 1] = id[thid + p1 - 1];
		//od[thid + p2 - 1] = id[thid + p2 - 1];
		//if (thid == n) { od[n - 1] = 0; }
		int t = od[thid +p1 - 1];//
		od[thid + p1 - 1] = od[thid + p2 - 1];
		od[thid + p2 - 1] += t;
		}


	
void init(int n, const int *hst_A){

	int _size = n*sizeof(int);
	hipMalloc((void**)&dev_A1, _size);
	hipMemcpy(dev_A1, hst_A, _size, hipMemcpyHostToDevice);


}


void scan(int n, int *odata, const int *idata) {
    // TODO

	//dev_A1,dev_B1
	int p1,p2;
	int num;
	if (n % 2 != 0)
	{
		num = ilog2ceil(n);
		num = pow(2, num);
	}
	else num = n;
	int *_idata = new int[num];
	init(num, idata);
	float ms=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&ms, start, stop);
	for (int d = 0; d <= ilog2ceil(num) - 1; d++){
		p1 = pow(2, d);
		p2 = pow(2, d + 1);
		Uscan << <1, 512 >> >(p1, p2, dev_A1);
	}
	put0 << <1, 1 >> >(dev_A1, num);
	for (int d = ilog2ceil(num) - 1; d >= 0; d--){
		p1 = pow(2, d);
		p2 = pow(2, d + 1);
		Dscan << <1, 512 >> >(p1, p2, dev_A1);
	
	}
	hipEventSynchronize(stop);

	hipEventElapsedTime(&ms, start, stop);
	printf("\t time of 3.1 efficient function1: %f ms\n", ms);
	hipMemcpy(odata, dev_A1, num* sizeof(int), hipMemcpyDeviceToHost);//destination,source,
	hipFree(dev_A1);

    printf("3.1\n");
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */ 
int *dev_idata;
int *dev_odata;
int *dev_indices;
int *dev_bool;
int *dev_boolb;

int compact(int n, int *odata, const int *idata) {
    // TODO
	int num;
	if (n % 2 != 0)
	{
		num = ilog2ceil(n);
		num = pow(2, num);
	}
	else num = n;

	int _size = num*sizeof(int);

	hipMalloc((void**)&dev_bool, _size);
	hipMalloc((void**)&dev_boolb, _size);
	hipMalloc((void**)&dev_odata, _size);
	hipMalloc((void**)&dev_idata, _size);
	hipMemcpy(dev_idata, idata, _size, hipMemcpyHostToDevice);

	int p1, p2;
	int hst;
	int last;
	//step 1

	Common::kernMapToBoolean <<< 1, n >>>(n, dev_bool, dev_idata);
	Common::kernMapToBoolean << < 1, n >> >(n, dev_boolb, dev_idata);//back_up
	//hipMemcpy(&hst, &dev_idata[6],sizeof(int), hipMemcpyDeviceToHost);
	//std::cout << hst;
   //Step 2 

	for (int d = 0; d <= ilog2ceil(num) - 1; d++){
			p1 = pow(2, d);
			p2 = pow(2, d + 1);
			Uscan <<<1, num >> >(p1, p2, dev_boolb);//change end to n
				}
	put0 <<<1, 1 >> >(dev_boolb, num);
	//hipMemcpy(&hst, &dev_idata[6], sizeof(int), hipMemcpyDeviceToHost);
	//std::cout << hst << "ss1";
	for (int d = ilog2ceil(num) - 1; d >= 0; d--){
			p1 = pow(2, d);
			p2 = pow(2, d + 1);
			Dscan <<<1, num >> >(p1, p2, dev_boolb);
			}


	//???????????my dev_idata changed its value here...have no idea why.
	hipMemcpy(dev_idata, idata, _size, hipMemcpyHostToDevice);
    ////???????????????????????/////////////
	//hipMemcpy(&hst, &dev_idata[6], sizeof(int), hipMemcpyDeviceToHost);
	//std::cout << hst << "ss2";
    //Step 3 : Scatter
	//hipMemcpy(&hst, &dev_idata[2],sizeof(int), hipMemcpyDeviceToHost);
	//std::cout << hst;
	hipMemcpy(&last, &(dev_boolb[num - 1]), sizeof(int), hipMemcpyDeviceToHost);
	//hipMalloc((void**)&dev_odata, last*sizeof(int));

	Common::kernScatter <<<1, num >> >(last, dev_odata, dev_idata, dev_bool, dev_boolb);

	hipMemcpy(odata, dev_odata, last*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("3.2\n");
    return last;
}

}
}
